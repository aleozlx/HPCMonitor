#include <iostream>
#include <chrono>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <hip/device_functions.h>

/*
nvcc -std=c++11 --compiler-options -Wall -O2 \
	-gencode arch=compute_30,code=sm_30 \
	-gencode arch=compute_35,code=sm_35 \
	-gencode arch=compute_50,code=sm_50 \
	-gencode arch=compute_52,code=sm_52 \
	-gencode arch=compute_60,code=sm_60 \
	-gencode arch=compute_61,code=sm_61 \
	-gencode arch=compute_61,code=compute_61 \
	-o build/vmem mem.cu && build/vmem
*/

// see also http://llvm.org/docs/CompileCudaWithLLVM.html
// ref: https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
//      http://docs.nvidia.com/cuda/cuda-runtime-api/#axzz4jNvlr4KG
//      https://developer.nvidia.com/cuda-code-samples

// typedef struct {
// 	boost::shared_ptr<boost::thread> handle;
// 	char *local_buffer;
// 	std::size_t ret;
// } thread_rc;

static std::size_t N_THREADS;
static std::size_t MEM_LIMIT = 2ul << 30;

void init(int argc, char *argv[]){
	int deviceCount; hipDeviceProp_t deviceProp;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess) std::exit(0);
	for (int device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&deviceProp, device);
		if (deviceProp.major == 9999) continue;
		if (device == 0) {
			printf("GPU processor count = %d\n", deviceProp.multiProcessorCount);
			printf("Threads per processor = %d\n", deviceProp.maxThreadsPerMultiProcessor);
			N_THREADS = deviceProp.maxThreadsPerMultiProcessor;
		}
	}

	N_THREADS = 32;

	if(argc > 1){
		int tmp = atoi(argv[1]);
		if(tmp > 0) MEM_LIMIT = static_cast<std::size_t>(tmp) << 20;
	}

	MEM_LIMIT /= N_THREADS;
	std::srand(std::time(0));
}

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernel_dummy(float *_buffer, std::size_t *_ret, std::size_t MEM_LIMIT){
	// std::size_t k = 0;
	// for(;k<((MEM_LIMIT/BLK_SZ)<<4);++k){
	// 	int random_block = std::rand()%(MEM_LIMIT-BLK_SZ);
	// 	std::memset(threads[i].local_buffer+random_block, 0xAA, BLK_SZ);
	// }

	_ret[threadIdx.x] = 1;
}

__global__ void kernel_sum_ret(std::size_t *_ret){
	extern __shared__ std::size_t sdata[];
	sdata[threadIdx.x] = _ret[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();

	for (std::size_t s = (blockDim.x>>1);s>0;s>>=1) {
		if(threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
		__syncthreads();
	}

	if (threadIdx.x == 0) _ret[blockIdx.x] = sdata[0];
}

void ramdom_write(std::size_t BLK_SZ){
	std::cout<<"Writing to "<<((MEM_LIMIT*N_THREADS)>>20)<<"MiB block with "<<
		N_THREADS<<" threads (BLK_SZ = "<<(BLK_SZ>>10)<<"KiB)..."<<std::endl;

	float *_buffer; hipMalloc(&_buffer, MEM_LIMIT*N_THREADS);
	std::size_t *_ret; hipMalloc(&_ret, sizeof(std::size_t)*N_THREADS);

	auto t1 = std::chrono::high_resolution_clock::now();
	kernel_dummy<<<1, N_THREADS>>>(_buffer, _ret, MEM_LIMIT);
	hipDeviceSynchronize();
	auto t2 = std::chrono::high_resolution_clock::now();
	kernel_sum_ret<<<1, N_THREADS, N_THREADS*sizeof(std::size_t)>>>(_ret);
	std::size_t total_size = 0;
	cudaCheck(hipMemcpy(&total_size, _ret, sizeof(std::size_t), hipMemcpyDeviceToHost));
	std::cout<<"total_size="<<total_size<<std::endl;
	std::chrono::duration<double, std::milli> time_ra = t2 - t1;
	std::cout<<"  "<<time_ra.count()<<"ms @ "<<
		static_cast<float>(total_size>>30)/time_ra.count()*1000.f<<"GiB/s"<<std::endl;
	hipFree(_buffer);
	hipFree(_ret);
}

int main(int argc, char *argv[]){
	init(argc, argv);

	ramdom_write(8 << 20);
	ramdom_write(1 << 20);
	ramdom_write(128 << 10);
	ramdom_write(16 << 10);
	ramdom_write(2 << 10);
	ramdom_write(1 << 10);

	// ramdom_read(512 << 10);
	// ramdom_read(8 << 10);
	// ramdom_read(4 << 10);
	// ramdom_read(2 << 10);
	// ramdom_read(1 << 10);
	// ramdom_read(256);

	return 0;
}
