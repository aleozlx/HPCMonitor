#include <iostream>
#include <chrono>
#include <vector>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/device_functions.h>

/*
nvcc -std=c++11 --compiler-options -Wall -O2 \
	-gencode arch=compute_30,code=sm_30 \
	-gencode arch=compute_35,code=sm_35 \
	-gencode arch=compute_50,code=sm_50 \
	-gencode arch=compute_52,code=sm_52 \
	-gencode arch=compute_60,code=sm_60 \
	-gencode arch=compute_61,code=sm_61 \
	-gencode arch=compute_61,code=compute_61 \
	-o build/vmem mem.cu
*/

/* see also http://llvm.org/docs/CompileCudaWithLLVM.html
clang++ -std=c++11 -Wall -O2 \
	--cuda-gpu-arch=sm_30 \
	--cuda-gpu-arch=sm_35 \
	--cuda-gpu-arch=sm_50 \
	--cuda-gpu-arch=sm_52 \
	--cuda-gpu-arch=sm_60 \
	--cuda-gpu-arch=sm_61 \
	-o build/vmem mem.cu \
	-L/usr/local/cuda/lib64 \
	-lcudart_static -ldl -lrt -pthread
*/

// ref: https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
//      http://docs.nvidia.com/cuda/cuda-runtime-api/#axzz4jNvlr4KG
//      https://developer.nvidia.com/cuda-code-samples

static std::size_t N_THREADS;
static std::size_t MEM_LIMIT = 2ul << 30;

void init(int argc, char *argv[]){
	int deviceCount; hipDeviceProp_t deviceProp;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess) std::exit(0);
	for (int device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&deviceProp, device);
		if (deviceProp.major == 9999) continue;
		if (device == 0) {
			printf("GPU processor count = %d\n", deviceProp.multiProcessorCount);
			printf("Threads per processor = %d\n", deviceProp.maxThreadsPerMultiProcessor);
			N_THREADS = deviceProp.maxThreadsPerMultiProcessor;
		}
	}

	N_THREADS = 32;

	if(argc > 1){
		int tmp = atoi(argv[1]);
		if(tmp > 0) MEM_LIMIT = static_cast<std::size_t>(tmp) << 20;
	}

	MEM_LIMIT /= N_THREADS;
}

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
	  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  if (abort) exit(code);
   }
}

__global__ void kernel_dummy(char *_buffer, std::size_t *_ret, std::size_t const MEM_LIMIT, std::size_t const BLK_SZ){
	hiprandState localState;
	hiprand_init(1234, threadIdx.x, 0, &localState);
	float *local_buffer = reinterpret_cast<float*>(_buffer+threadIdx.x*MEM_LIMIT);
	float const val = 2.71828f;
	std::size_t k = 0;
	for(;k<((MEM_LIMIT/BLK_SZ));++k){
		std::size_t random_block = static_cast<std::size_t>(hiprand_uniform(&localState) * ((MEM_LIMIT-BLK_SZ)/sizeof(float)));
		for(std::size_t p = 0;p<(MEM_LIMIT/sizeof(float));++p) // TODO BLK_SZ
			(local_buffer+random_block)[p] = val;
	}

	_ret[threadIdx.x] = k*BLK_SZ;
}

__global__ void kernel_sum_ret(std::size_t *_ret){
	extern __shared__ std::size_t sdata[];
	sdata[threadIdx.x] = _ret[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();

	for (std::size_t s = (blockDim.x>>1);s>0;s>>=1) {
		if(threadIdx.x < s) sdata[threadIdx.x] += sdata[threadIdx.x + s];
		__syncthreads();
	}

	if (threadIdx.x == 0) _ret[blockIdx.x] = sdata[0];
}

void ramdom_write(std::size_t BLK_SZ){
	std::cout<<"Writing to "<<((MEM_LIMIT*N_THREADS)>>20)<<"MiB block with "<<
		N_THREADS<<" threads (BLK_SZ = "<<(BLK_SZ>>10)<<"KiB)..."<<std::endl;

	char *_buffer; hipMalloc(&_buffer, MEM_LIMIT*N_THREADS);
	std::size_t *_ret; hipMalloc(&_ret, sizeof(std::size_t)*N_THREADS);

	auto t1 = std::chrono::high_resolution_clock::now();
	kernel_dummy<<<1, N_THREADS>>>(_buffer, _ret, MEM_LIMIT, BLK_SZ);
	hipDeviceSynchronize();
	auto t2 = std::chrono::high_resolution_clock::now();
	kernel_sum_ret<<<1, N_THREADS, N_THREADS*sizeof(std::size_t)>>>(_ret);
	std::size_t total_size = 0;
	cudaCheck(hipMemcpy(&total_size, _ret, sizeof(std::size_t), hipMemcpyDeviceToHost));
	std::cout<<"total_size="<<total_size<<std::endl;
	std::chrono::duration<double, std::milli> time_ra = t2 - t1;
	std::cout<<"  "<<time_ra.count()<<"ms @ "<<
		static_cast<float>(total_size>>30)/time_ra.count()*1000.f<<"GiB/s"<<std::endl;
	hipFree(_buffer);
	hipFree(_ret);
}

int main(int argc, char *argv[]){
	init(argc, argv);

	ramdom_write(8 << 20);
	ramdom_write(1 << 20);
	// ramdom_write(128 << 10);
	// ramdom_write(16 << 10);
	// ramdom_write(2 << 10);
	// ramdom_write(1 << 10);

	// ramdom_read(512 << 10);
	// ramdom_read(8 << 10);
	// ramdom_read(4 << 10);
	// ramdom_read(2 << 10);
	// ramdom_read(1 << 10);
	// ramdom_read(256);

	return 0;
}
